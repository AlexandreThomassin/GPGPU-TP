#include "hip/hip_runtime.h"
#include "ann.h"
#include "matrix.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <float.h>
#include <stdbool.h>
#include <stdint.h>

double normalRand(double mu, double sigma);
void init_weight(matrix_t* w, unsigned nneurones_prev);
void print_layer(layer_t *layer);

// const int N = 1024 * 1024;
// const int threadsPerBlock = 256;
// const int blocksPerGrid = (N+threadsPerBlock-1) / threadsPerBlock ;

double normalRand(double mu, double sigma)
{
	const double epsilon = DBL_MIN;
	const double two_pi = 2.0*M_PI;
    static bool generate;
    static double z1;

	generate = !generate;

	if (!generate)
	   return z1 * sigma + mu;

	double u1, u2;
	do
	 {
	   u1 = (double) rand() / RAND_MAX;
	   u2 = (double) rand() / RAND_MAX;
	 }
	while ( u1 <= epsilon );

	double z0;
	z0 = sqrt(-2.0 * log(u1)) * cos(two_pi * u2);
	z1 = sqrt(-2.0 * log(u1)) * sin(two_pi * u2);
	return z0 * sigma + mu;
}

void init_weight(matrix_t* w, unsigned nneurones_prev)
{
    double *m = (double*) malloc(w->rows * w->columns * sizeof(double));
    for (int idx = 0; idx < w->columns * w->rows; idx ++)
    {
        m[idx] =  normalRand(0, 1 / sqrt(nneurones_prev));
    }
    hipMemcpy(w->m, m, w->rows * w->columns * sizeof(double), hipMemcpyHostToDevice);
}

ann_t * create_ann_kernel(double alpha, unsigned minibatch_size, unsigned number_of_layers, unsigned* nneurons_per_layer)
{
    ann_t *nn;

    hipMalloc((void **) &nn, sizeof(ann_t));

    layer_t **layers = (layer_t **)malloc(number_of_layers * sizeof(layer_t*));

    hipMemcpy(&(nn->number_of_layers), &number_of_layers, sizeof(unsigned), hipMemcpyHostToDevice);
    hipMemcpy(&(nn->alpha), &alpha, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(&(nn->minibatch_size), &minibatch_size, sizeof(unsigned), hipMemcpyHostToDevice);

    layer_t *layer0 = create_layer_kernel(0, nneurons_per_layer[0], minibatch_size, minibatch_size);
    layers[0] = layer0;

    for (int l = 1; l < number_of_layers; l++)
    {
        layer_t *layerl = create_layer_kernel(l, nneurons_per_layer[l], nneurons_per_layer[l-1], minibatch_size);
        layers[l] = layerl;
    }

    
    layer_t **d_layers;
    hipMalloc((void **) &d_layers, number_of_layers * sizeof(layer_t*));
    hipMemcpy(d_layers, layers, number_of_layers * sizeof(layer_t), hipMemcpyHostToDevice);

    hipMemcpy(&(nn->layers), &d_layers, sizeof(layer_t**), hipMemcpyHostToDevice);

    return nn;
}

ann_t * create_ann(double alpha, unsigned minibatch_size, unsigned number_of_layers, unsigned* nneurons_per_layer)
{
    ann_t * nn = (ann_t *)malloc(sizeof(ann_t));

    nn->layers = (layer_t **)malloc(number_of_layers * sizeof(layer_t *));
    nn->number_of_layers = number_of_layers;
    nn->alpha = alpha;
    nn->minibatch_size = minibatch_size;

    nn->layers[0] = create_layer(0, nneurons_per_layer[0], minibatch_size, minibatch_size);
    for (int l = 1; l < number_of_layers; l++)
    {
        nn->layers[l] = create_layer(l, nneurons_per_layer[l], nneurons_per_layer[l-1], minibatch_size);
    }

    return nn;
}

layer_t * create_layer_kernel(unsigned layer_number, unsigned number_of_neurons, unsigned nneurons_previous_layer, unsigned minibatch_size)
{
    layer_t *layer;
    hipMalloc((void **) &layer, sizeof(layer_t));

    hipMemcpy(&(layer->number_of_neurons), &number_of_neurons, sizeof(unsigned), hipMemcpyHostToDevice);
    hipMemcpy(&(layer->minibatch_size), &minibatch_size, sizeof(unsigned), hipMemcpyHostToDevice);
    
    matrix_t *activations = alloc_matrix_kernel(number_of_neurons, minibatch_size);
    matrix_t *z = alloc_matrix_kernel(number_of_neurons, minibatch_size);
    matrix_t *delta = alloc_matrix_kernel(number_of_neurons, minibatch_size);
    matrix_t *weights = alloc_matrix_kernel(number_of_neurons, nneurons_previous_layer);    
    matrix_t *biases = alloc_matrix_kernel(number_of_neurons, 1);

    hipMemcpy(&(layer->activations), &activations, sizeof(matrix_t*), hipMemcpyHostToDevice);
    hipMemcpy(&(layer->z), &z, sizeof(matrix_t*), hipMemcpyHostToDevice);
    hipMemcpy(&(layer->delta), &delta, sizeof(matrix_t*), hipMemcpyHostToDevice);
    hipMemcpy(&(layer->biases), &biases, sizeof(matrix_t*), hipMemcpyHostToDevice);

    if (layer_number > 0)
    {
        matrix_t *w = (matrix_t*)malloc(sizeof(matrix_t));
        init_weight(w, nneurons_previous_layer);
        hipMemcpy(weights, w, sizeof(matrix_t), hipMemcpyHostToDevice);
    }

    hipMemcpy(&(layer->weights), &weights, sizeof(matrix_t*), hipMemcpyHostToDevice);
    return layer;
}

layer_t * create_layer(unsigned layer_number, unsigned number_of_neurons, unsigned nneurons_previous_layer, unsigned minibatch_size)
{
    layer_t * layer = (layer_t*) malloc(sizeof(layer_t));

    layer->number_of_neurons = number_of_neurons;
    layer->minibatch_size = minibatch_size;    
    layer->activations = alloc_matrix_kernel(number_of_neurons, minibatch_size);
    layer->z = alloc_matrix_kernel(number_of_neurons, minibatch_size);
    layer->delta = alloc_matrix_kernel(number_of_neurons, minibatch_size);
    layer->weights = alloc_matrix_kernel(number_of_neurons, nneurons_previous_layer);    
    layer->biases = alloc_matrix_kernel(number_of_neurons, 1);

    if (layer_number > 0)
    {
        init_weight(layer->weights, nneurons_previous_layer);
    }

    return layer;
}

void set_input(ann_t *nn, matrix_t* input){
    matrix_memcpy(nn->layers[0]->activations, input);
}

void print_layer(layer_t *layer)
{
    printf("-- neurons:%d, minibatch size:%d\n", layer->number_of_neurons, layer->minibatch_size);

    printf(">> Weighted inputs --\n");
    print_matrix(layer->z, true);
    printf(">> Activations --\n");
    print_matrix(layer->activations, true);
    
    printf(">> Weights --\n");
    print_matrix(layer->weights, true);
    printf(">> Biases --\n");
    print_matrix(layer->biases, true);

    printf(">> Delta --\n");
    print_matrix(layer->delta, true);
    
}

void print_nn(ann_t *nn)
{
    printf("ANN -- nlayers:%d, alpha:%lf, minibatch size: %d\n", nn->number_of_layers, nn->alpha, nn->minibatch_size);
    for (int l = 0; l < nn->number_of_layers; l++)
    {
        printf("Layer %d ", l);
        print_layer(nn->layers[l]);
    }
}

void forward(ann_t *nn)
{

    for (int l = 1; l < nn->number_of_layers; l++)
    {
        matrix_t *z1 = alloc_matrix_kernel(nn->layers[l]->number_of_neurons, nn->minibatch_size);
        matrix_t *z2 = alloc_matrix_kernel(nn->layers[l]->number_of_neurons, nn->minibatch_size);
        matrix_t *one = alloc_matrix_kernel(1, nn->minibatch_size);

        // double *m = (double*) malloc(one->columns * one->rows * sizeof(double));
        // for (int idx = 0; idx < one->columns*one->rows; idx++)
        //     m[idx] = 1.0;

        // hipMemcpy(one->m, m, one->rows * one->columns * sizeof(double), hipMemcpyHostToDevice);
        hipMemset(one->m, 1, one->rows * one->columns * sizeof(double));
        
        dim3 blockDim(16, 16);
        dim3 gridDim(ceil(((double)nn->layers[l]->weights->columns) / blockDim.x), ceil(((double)nn->layers[l-1]->activations->rows) / blockDim.y));
        matrix_dot_kernel<<<gridDim, blockDim>>>(nn->layers[l]->weights->m, nn->layers[l-1]->activations->m, z1->m, nn->layers[l]->weights->rows, nn->layers[l]->weights->columns, nn->layers[l-1]->activations->rows, nn->layers[l-1]->activations->columns); // z1 <- w^l x a^(l-1)
        hipDeviceSynchronize();

        dim3 gridDim2(ceil(((double)nn->layers[l]->biases->columns) / blockDim.x), ceil(((double)one->rows) / blockDim.y));
        matrix_dot_kernel<<<gridDim2, blockDim>>>(nn->layers[l]->biases->m, one->m, z2->m, nn->layers[l]->biases->rows, nn->layers[l]->biases->columns, one->rows, one->columns); // z2 <- b^l x 1        
        hipDeviceSynchronize();

        dim3 gridDim3(ceil(((double)nn->layers[l]->z->columns) / blockDim.x), ceil(((double)nn->layers[l]->z->rows) / blockDim.y));
        matrix_sum_kernel<<<gridDim3, blockDim>>>(z1->m, z2->m, nn->layers[l]->z->m, nn->layers[l]->z->rows, nn->layers[l]->z->columns); // z^l <- z1 + z2 <=> z^l <- w^l x a^(l-1) + b^l x 1  
        hipDeviceSynchronize();

        matrix_function_kernel_sig<<<gridDim3, blockDim>>>(nn->layers[l]->z->m, nn->layers[l]->activations->m, nn->layers[l]->activations->rows, nn->layers[l]->activations->columns);
        hipDeviceSynchronize();


        destroy_matrix_kernel(z1);
        destroy_matrix_kernel(z2);
        destroy_matrix_kernel(one);

    }
}

void backward(ann_t *nn, matrix_t *y)
{
    unsigned L = nn->number_of_layers-1;

    matrix_t *dfzL = alloc_matrix_kernel(nn->layers[L]->number_of_neurons, nn->minibatch_size);

    dim3 blockDim(16, 16);
    dim3 gridDim(ceil(((double)nn->layers[L]->activations->columns) / blockDim.x), ceil(((double)nn->layers[L]->activations->rows) / blockDim.y));

    matrix_minus_kernel<<<gridDim, blockDim>>>(nn->layers[L]->activations->m, y->m, nn->layers[L]->delta->m, y->rows, y->columns);  // delta^(L) = (a^L - y)

    matrix_function_kernel_dsig<<<gridDim, blockDim>>>(nn->layers[L]->z->m, dfzL->m, nn->layers[L]->z->rows, nn->layers[L]->z->columns); // f'(z^(L))

    hadamard_product_kernel<<<gridDim, blockDim>>>(nn->layers[L]->delta->m, dfzL->m, nn->layers[L]->delta->m, nn->layers[L]->delta->rows, nn->layers[L]->delta->columns); // delta^(L) = (a^L - y) o f'(z^(L))

    destroy_matrix_kernel(dfzL);

    for (int l = L; l > 1; l--)
    {
        matrix_t *tw, *delta_tmp, *dfz;
        tw = alloc_matrix_kernel(nn->layers[l-1]->number_of_neurons, nn->layers[l]->number_of_neurons);
        delta_tmp = alloc_matrix_kernel(nn->layers[l-1]->number_of_neurons, nn->minibatch_size);
        dfz = alloc_matrix_kernel(nn->layers[l-1]->number_of_neurons, nn->minibatch_size);

        dim3 gridDim2(ceil(((double)tw->rows) / blockDim.x), ceil(((double)tw->columns) / blockDim.y));
        matrix_transpose_kernel<<<gridDim2, blockDim>>>(nn->layers[l]->weights->m, tw->m, tw->columns, tw->rows); // (w^l)T      

        dim3 gridDim3(ceil(((double)tw->columns) / blockDim.x), ceil(((double)delta_tmp->rows) / blockDim.y));
        matrix_dot_kernel<<<gridDim3, blockDim>>>(tw->m, nn->layers[l]->delta->m, delta_tmp->m, tw->rows, tw->columns, nn->layers[l]->delta->rows, nn->layers[l]->delta->columns); // (w^l)T x delta^l

        dim3 gridDim4(ceil(((double)dfz->columns) / blockDim.x), ceil(((double)dfz->rows) / blockDim.y));
        matrix_function_kernel_dsig<<<gridDim4, blockDim>>>(nn->layers[l-1]->z->m, dfz->m, dfz->rows, dfz->columns); // f'(z^(l-1))
        hadamard_product_kernel<<<gridDim4, blockDim>>>(delta_tmp->m, dfz->m, nn->layers[l-1]->delta->m, dfz->rows, dfz->columns); // delta^(l-1) = (w^l)T x delta^l o f'(z^(l-1))

        destroy_matrix_kernel(tw);
        destroy_matrix_kernel(delta_tmp);
        destroy_matrix_kernel(dfz);
    }

    for (int l = 1; l < nn->number_of_layers; l++)
    {
        matrix_t *w1, *ta;
        w1 = alloc_matrix_kernel(nn->layers[l]->number_of_neurons, nn->layers[l-1]->number_of_neurons);
        ta = alloc_matrix_kernel(nn->minibatch_size, nn->layers[l-1]->number_of_neurons);
        
        dim3 gridDim5(ceil(((double)ta->rows) / blockDim.x), ceil(((double)ta->columns) / blockDim.y));
        matrix_transpose_kernel<<<gridDim5, blockDim>>>(nn->layers[l-1]->activations->m, ta->m, ta->columns, ta->rows); // ta <- (a^(l-1))^T

        dim3 gridDim6(ceil(((double) w1->columns) / blockDim.x), ceil(((double)w1->rows) / blockDim.y));
        matrix_dot_kernel<<<gridDim6, blockDim>>>(nn->layers[l]->delta->m, ta->m, w1->m, nn->layers[l]->delta->rows, nn->layers[l]->delta->columns, ta->rows, ta->columns); // w1 <- delta^l x (a^(l-1))^T

        matrix_scalar_kernel<<<gridDim6, blockDim>>>(w1->m, nn->alpha / nn->minibatch_size, w1->m, w1->rows, w1->columns); // w1 <- alpha /m . delta^l x (a^(l-1))^T
        matrix_minus_kernel<<<gridDim6, blockDim>>>(nn->layers[l]->weights->m, w1->m, nn->layers[l]->weights->m, w1->rows, w1->columns); // w^l <- w^l - alpha /m . delta^l x (a^(l-1))^T

        destroy_matrix_kernel(w1);
        destroy_matrix_kernel(ta);

        matrix_t *one, *b1;
        b1 = alloc_matrix_kernel(nn->layers[l]->number_of_neurons, 1);
        one = alloc_matrix_kernel(nn->minibatch_size, 1);
        // double *m = (double*)malloc(one->columns * one->rows * sizeof(double));
        // for (int idx = 0; idx < one->columns*one->rows; idx++)
        //     m[idx] = 1.0;

        // hipMemcpy(one->m, m, one->rows * one->columns * sizeof(double), hipMemcpyHostToDevice);

        // free(m);
        hipMemset(one->m, 1, one->rows * one->columns * sizeof(double));


        dim3 gridDim7(ceil(((double) b1->columns) / blockDim.x), ceil(((double)b1->rows) / blockDim.y));
        matrix_dot_kernel<<<gridDim7, blockDim>>>(nn->layers[l]->delta->m, one->m, b1->m, nn->layers[l]->delta->rows, nn->layers[l]->delta->columns, one->rows, one->columns); // b1 <- delta^l x 1^T
        matrix_scalar_kernel<<<gridDim7, blockDim>>>(b1->m,  nn->alpha / nn->minibatch_size, b1->m, b1->rows, b1->columns); // b1 <- alpha / m . delta^l x 1^T
        matrix_minus_kernel<<<gridDim7, blockDim>>>(nn->layers[l]->biases->m, b1->m, nn->layers[l]->biases->m, b1->rows, b1->columns); // b^l = b^l - alpha / m . delta^l x 1^T
        
        // destroy_matrix_kernel(one);
        // destroy_matrix_kernel(b1);
    }
}

void forward_CPU(ann_t *nn, double (*activation_function)(double))
{

    for (int l = 1; l < nn->number_of_layers; l++)
    {
        matrix_t *z1 = alloc_matrix(nn->layers[l]->number_of_neurons, nn->minibatch_size);
        matrix_t *z2 = alloc_matrix(nn->layers[l]->number_of_neurons, nn->minibatch_size);
        matrix_t *one = alloc_matrix(1, nn->minibatch_size);
        for (int idx = 0; idx < one->columns*one->rows; idx++)
            one->m[idx] = 1.0;

        matrix_dot(nn->layers[l]->weights, nn->layers[l-1]->activations, z1); // z1 <- w^l x a^(l-1)
        matrix_dot(nn->layers[l]->biases, one, z2); // z2 <- b^l x 1        
        matrix_sum(z1, z2, nn->layers[l]->z); // z^l <- z1 + z2 <=> z^l <- w^l x a^(l-1) + b^l x 1      

        matrix_function(nn->layers[l]->z, activation_function, nn->layers[l]->activations); // a^l = f(z^l)
     
        destroy_matrix_kernel(z1);
        destroy_matrix_kernel(z2);
        destroy_matrix_kernel(one);
    }
}